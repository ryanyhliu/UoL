#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>


__global__ void vectorArithmetic(float *z, const float *x, const float *y, float A, int num) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < num) {
        z[i] = A * x[i] + y[i];
    }
}


int main(void){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);



	//declaration of host variables
	const int num = 1000000;
	float *z, *x, *y;


	
	float A = 34;
	
	//initialising host variables
	z = (float*) malloc(num * sizeof(float));
	x = (float*) malloc(num * sizeof(float));
	y = (float*) malloc(num * sizeof(float));
	
	for(int i = 0; i < num; i++){
		x[i] = i;
		y[i] = 7 * i;
	}

	// //task to be performed by GPU. Don't forget to add global
	// for(int i = 0; i < num; i++){
	// 	z[i] = A * x[i] + y[i];
	// }
	


	float *d_x, *d_y, *d_z;
	hipMalloc((void **)&d_x, num * sizeof(float));
	hipMalloc((void **)&d_y, num * sizeof(float));
	hipMalloc((void **)&d_z, num * sizeof(float));

	hipMemcpy(d_x, x, num * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, num * sizeof(float), hipMemcpyHostToDevice);

	int blockSize = 512; // 可以根据需要调整
	int numBlocks = (num + blockSize - 1) / blockSize;

	hipEventRecord(start);
	vectorArithmetic<<<numBlocks, blockSize>>>(d_z, d_x, d_y, A, num);
	hipEventRecord(stop);

	hipMemcpy(z, d_z, num * sizeof(float), hipMemcpyDeviceToHost);

	//postprocessing: output to terminal
	for(int j = 0; j < num; j++){
		printf("%f ", z[j]);
	}

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Elapsed time: %f ms\n", milliseconds);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	
	return 0;
}
