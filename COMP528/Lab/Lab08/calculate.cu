#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define MAX_NUM 100

__global__
void printNums(){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index <= MAX_NUM){
        int square = index * index;
        if (square <= MAX_NUM){
            printf("%d ^ 2 = %d\n", index, square);
        }
    }
}

int main(){
    int threadsPerBlk = 256;
    int blocks = (MAX_NUM + threadsPerBlk - 1) / threadsPerBlk;

    printNums<<<blocks, threadsPerBlk>>>();
    hipDeviceSynchronize();

    return 0;
}



