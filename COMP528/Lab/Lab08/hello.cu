
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__
void helloKernel(int N)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  if (i<N) {
    printf("thread %d of block %d (dim: %d): iter %d of %d\n", threadIdx.x, blockIdx.x, blockDim.x, i, N);
  }
}


int main(void) {
  int N=20,numGPUs;
  int blks=5;
  int threadsPerBlk=4;

  /* check have GPU else quit */
  hipGetDeviceCount(&numGPUs);
  printf("GPU nums: %d \n", numGPUs);


  if (numGPUs >= 1 ) {

    printf("hello on CPU\n");

    /* call GPU kernel  */
    helloKernel<<<blks,threadsPerBlk>>> (N);
    // cudaDeviceSynchronize();


    printf("hello again from CPU\n");


  }else {
    printf("no GPU present\n");
  }

}
