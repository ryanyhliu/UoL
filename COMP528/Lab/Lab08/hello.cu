
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__
void helloKernel(int N)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  if (i<N) {
    printf("thread %d of block %d (dim: %d): iter %d of %d\n", threadIdx.x, blockIdx.x, blockDim.x, i, N);
  }
}


int main(void) {
  int N=10,numGPUs;
  int blks=4;
  int threadsPerBlk=3;

  /* check have GPU else quit */
  hipGetDeviceCount(&numGPUs);
  if (numGPUs >= 1 ) {

  printf("hello on CPU\n");

  /* call GPU kernel  */
  helloKernel<<<blks,threadsPerBlk>>> (N);

  printf("hello again from CPU\n");


}

  else {
   printf("no GPU present\n");
}

}
